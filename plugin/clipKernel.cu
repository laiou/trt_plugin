#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <clipKernel.h>

template <typename T>
//__forceinline__指定编译器内联函数
__device__ __forceinline__ const T& min(const T& a, const T& b)
{
    return (a > b) ? b : a;
}

template <typename T>
__device__ __forceinline__ const T& max(const T& a, const T& b)
{
    return (a > b) ? a : b;
}

template <typename T, unsigned nthdsPerCTA>
//__launch_bounds__指定一个线程块上运行的线程数量，从而然后编译器做寄存器分配上的优化
__launch_bounds__(nthdsPerCTA)
    __global__ void clipKernel(
        int n,
        const T clipMin,
        const T clipMax,
        const T* input,
        T* output)
{
    //计算相应的线程id，然后执行clip操作
    for (int i = blockIdx.x * nthdsPerCTA + threadIdx.x; i < n; i += gridDim.x * nthdsPerCTA)
    {
        output[i] = min<T>(max<T>(input[i], clipMin), clipMax);
    }
}
//执行clip的推理过程
int clipInference(
    hipStream_t stream,
    int n,
    float clipMin,
    float clipMax,
    const void* input,
    void* output)
{
    //定义相应的blocksize
    const int blockSize = 512;
    //以及相应的grid维度，保证划分是足够的并且gridSize是一个整数
    const int gridSize = (n + blockSize - 1) / blockSize;
    //clipKernel参考本文件的实现
    //clipKernel是一个c++中的模板，通过<float,blockSize>进行实例化
    clipKernel<float, blockSize><<<gridSize, blockSize, 0, stream>>>(n, clipMin, clipMax,
                                                 static_cast<const float*>(input),
                                                 static_cast<float*>(output));
    return 0;
}
